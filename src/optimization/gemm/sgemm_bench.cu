#include "hip/hip_runtime.h"
#include "gflags/gflags.h"
#include "naive.cuh"
#include "sgemm.cuh"
#include "sgemm_buffer.cuh"
#include "sgemm_reg_buffer.cuh"
#include "sgemm_v3.cuh"
#include "utils.cuh"
#include "utils.h"
#include "gtest/gtest.h"
#include <hipblas.h>

using FloatVector = std::vector<float>;

using SGemmFunc = std::function<void(const float *A, const float *B, float *C,
                                     int M, int N, int K)>;

class CUSgemmBench : public ::testing::Test {

public:
  void Verify(SGemmFunc func) {
    loops = 1;
    BenchFunc(func, true);
  }
  void BenchBlas() { BenchFunc(nullptr, false, true); }
  double Bench(SGemmFunc func) { return BenchFunc(func, false, false); }

protected:
  double BenchFunc(SGemmFunc func, bool verify = false, bool blas = false) {
    int ASIZE = m * k, BSIZE = k * n, CSIZE = m * n;
    int ABYTES = ASIZE * sizeof(float);
    int BBYTES = BSIZE * sizeof(float);
    int CBYTES = CSIZE * sizeof(float);

    std::vector<float> times;
    hipblasHandle_t blas_handle;
    CUBLAS_ERROR_CHECK(hipblasCreate(&blas_handle));
    for (int i = 0; i < loops; i++) {
      // Allocate host memory
      FloatVector hA = CreateData<float>(ASIZE, 0, 6);
      FloatVector hB = CreateData<float>(BSIZE, 0, 6);
      FloatVector hC(CSIZE, 0);

      // Allocate device memory
      float *dA, *dB, *dC;
      CUDA_ERROR_CHECK(hipMalloc(&dA, ABYTES));
      CUDA_ERROR_CHECK(hipMalloc(&dB, BBYTES));
      CUDA_ERROR_CHECK(hipMalloc(&dC, CBYTES));
      // Copy memory from host to device
      CUDA_ERROR_CHECK(
          hipMemcpy(dA, hA.data(), ABYTES, hipMemcpyHostToDevice));
      CUDA_ERROR_CHECK(
          hipMemcpy(dB, hB.data(), BBYTES, hipMemcpyHostToDevice));

      float time_ms = 0;
      hipEvent_t start, stop;
      CUDA_ERROR_CHECK(hipEventCreate(&start));
      CUDA_ERROR_CHECK(hipEventCreate(&stop));

      checkCudaErrors(hipEventRecord(start));
      if (blas) {
        float alpha = 1.0;
        float beta = 0;
        hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dB,
                    n, dA, k, &beta, dC, n);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
          printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
      } else {
        func(dA, dB, dC, m, n, k);
      }
      CUDA_ERROR_CHECK(hipEventRecord(stop));
      CUDA_ERROR_CHECK(hipEventSynchronize(stop));
      CUDA_ERROR_CHECK(hipEventElapsedTime(&time_ms, start, stop));

      // Copy memory from devie to host
      CUDA_ERROR_CHECK(
          hipMemcpy(hC.data(), dC, CBYTES, hipMemcpyDeviceToHost));
      hipDeviceSynchronize();

      if (verify) {
        FloatVector groundTruth(CSIZE, 0);
        RawMatmul(hA.data(), hB.data(), groundTruth.data(), m, n, k);
        FloatsCompare(hC.data(), groundTruth.data(), m * n);
      }

      // Free device memory
      hipFree(dA);
      hipFree(dB);
      hipFree(dC);

      times.push_back(time_ms);
    }
    hipblasDestroy(blas_handle);
    double gflops = (double)2 * m * n * k / (Average(times) * 1e6);
    std::cout << "Average Time: " << Average(times) << " gflops:" << gflops
              << std::endl;
    return gflops;
  }
  int m, n, k;
  int loops = 12;
};

class CUGemmMNK : public ::CUSgemmBench {
public:
  void Bench(SGemmFunc func) {
    table.SetHeads({"MNK", "GFlops"});
    for (int i = start; i < end; i += stride) {
      this->m = i;
      this->n = i;
      this->k = i;
      double glops = CUSgemmBench::Bench(func);
      table.AddRow({(float)i, (float)(glops)});
    }
    table.Print();
    table.ExportToCSV(
        "/media/tyger/linux_ssd/codes/cxx_test/cuda_lab/data/sgemm.csv");
  }

public:
  int start = 256, stride = 256, end = 4352;

private:
  Table2D table;
};

TEST_F(CUSgemmBench, v3) {
  m = 512, n = 512, k = 512;
  constexpr int BLOCK_SIZE_M = 128;
  constexpr int BLOCK_SIZE_K = 8;
  constexpr int BLOCK_SIZE_N = 128;
  constexpr int THREAD_SIZE_X = 8;
  constexpr int THREAD_SIZE_Y = 8;
  constexpr bool ENABLE_DOUBLE_BUFFER = false;

  Bench(SGemmV3<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_X,
                THREAD_SIZE_Y, ENABLE_DOUBLE_BUFFER>);
}

TEST_F(CUSgemmBench, v0) {
  m = 1024, n = 4096, k = 4096;

  constexpr int WY = 4;
  constexpr int WX = 8;

  constexpr int NR = 8;
  constexpr int MR = 8;

  constexpr int BX = WX * 2;
  constexpr int BY = WY * 4;

  constexpr int MC = MR * BY;
  constexpr int NC = NR * BX;

  constexpr int KC = 16;

  constexpr int BLOCK_SIZE_M = 128;
  constexpr int BLOCK_SIZE_K = 8;
  constexpr int BLOCK_SIZE_N = 128;
  constexpr int THREAD_SIZE_X = 8;
  constexpr int THREAD_SIZE_Y = 8;
  constexpr bool ENABLE_DOUBLE_BUFFER = false;

  loops = 1;
  Bench(CudaSGemm<MC, KC, NC, MR, NR, WY, WX>);
  // Bench(SGemmV3<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_X,
  //               THREAD_SIZE_Y, ENABLE_DOUBLE_BUFFER>);

  // Bench(CudaSGemmRegBuffer<MC, KC, NC, MR, NR, WY, WX>);
  Bench(CudaSGemmDoubleBuffer<MC, KC, NC, MR, NR, WY, WX>);
  // BenchBlas();
}

// TEST_F(CUGemmMNK, naive) { Bench(CublasSgemm); }