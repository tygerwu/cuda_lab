#include "hip/hip_runtime.h"
#include "gflags/gflags.h"
#include "hgemm_tc_v0.cuh"
#include "naive.cuh"
#include "utils.cuh"
#include "utils.h"
#include "gtest/gtest.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>

using HGemmFunc = std::function<void(const half *A, const half *B, half *C,
                                     int M, int N, int K)>;

template <hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP>
static void CublasF16F16Gemm(const half *a, const half *b, half *c, int M,
                             int N, int K) {

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  half alpha = 1.0;
  half beta = 0.0;
  CUBLAS_ERROR_CHECK(hipblasGemmEx(
      handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b, HIP_R_16F, N, a,
      HIP_R_16F, K, &beta, c, HIP_R_16F, N, HIPBLAS_COMPUTE_16F, algo));
  hipblasDestroy(handle);
}

class CUHGemmBench : public ::testing::Test {

public:
  void Verify(HGemmFunc func) {
    loops = 1;
    BenchFunc(func, true);
  }
  void BenchBlas() { BenchFunc(nullptr, false, true); }
  double Bench(HGemmFunc func) { return BenchFunc(func, false, false); }

protected:
  double BenchFunc(HGemmFunc func, bool verify = false, bool blas = false) {
    int ASIZE = m * k, BSIZE = k * n, CSIZE = m * n;
    int ABYTES = ASIZE * sizeof(half);
    int BBYTES = BSIZE * sizeof(half);
    int CBYTES = CSIZE * sizeof(half);

    std::vector<float> times;
    for (int i = 0; i < loops; i++) {
      // Allocate fp32 host memory
      FloatVector fp32_hA = CreateData<float>(ASIZE, 0, 6);
      FloatVector fp32_hB = CreateData<float>(BSIZE, 0, 6);
      FloatVector fp32_hC(CSIZE, 0);

      // Fp16 host memory
      std::vector<half> hA = Convert<float, half>(fp32_hA);
      std::vector<half> hB = Convert<float, half>(fp32_hB);
      std::vector<half> hC(CSIZE, 0);

      // Allocate device memory
      half *dA, *dB, *dC;
      CUDA_ERROR_CHECK(hipMalloc(&dA, ABYTES));
      CUDA_ERROR_CHECK(hipMalloc(&dB, BBYTES));
      CUDA_ERROR_CHECK(hipMalloc(&dC, CBYTES));

      // Copy memory from host to device
      CUDA_ERROR_CHECK(
          hipMemcpy(dA, hA.data(), ABYTES, hipMemcpyHostToDevice));
      CUDA_ERROR_CHECK(
          hipMemcpy(dB, hB.data(), BBYTES, hipMemcpyHostToDevice));

      float time_ms = 0;
      hipEvent_t start, stop;
      CUDA_ERROR_CHECK(hipEventCreate(&start));
      CUDA_ERROR_CHECK(hipEventCreate(&stop));
      CUDA_ERROR_CHECK(hipEventRecord(start));

      func(dA, dB, dC, m, n, k);
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
      }

      CUDA_ERROR_CHECK(hipEventRecord(stop));
      CUDA_ERROR_CHECK(hipEventSynchronize(stop));
      CUDA_ERROR_CHECK(hipEventElapsedTime(&time_ms, start, stop));

      // Copy memory from devie to host
      CUDA_ERROR_CHECK(
          hipMemcpy(hC.data(), dC, CBYTES, hipMemcpyDeviceToHost));
      hipDeviceSynchronize();

      // Fp16 to Fp 32
      fp32_hC = Convert<half, float>(hC);

      if (verify) {
        FloatVector groundTruth(CSIZE, 0);
        RawMatmul(fp32_hA.data(), fp32_hB.data(), groundTruth.data(), m, n, k);
        FloatsCompare(fp32_hC.data(), groundTruth.data(), m * n);
      }

      // Free device memory
      hipFree(dA);
      hipFree(dB);
      hipFree(dC);

      times.push_back(time_ms);
    }
    double gflops = (double)2 * m * n * k / (Average(times) * 1e6);
    std::cout << "Average Time: " << Average(times) << " gflops:" << gflops
              << std::endl;
    return gflops;
  }
  int m, n, k;
  int loops = 12;
};
