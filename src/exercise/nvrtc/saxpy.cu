#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "gtest/gtest.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hiprtc.h>
#include <string>
#define NUM_THREADS 128
#define NUM_BLOCKS 32

const char *saxpy = "\n\
extern \"C\" __global__ void saxpy(float a, float *x, float *y, float *out, size_t n) { \n\
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x; \n\
  if (tid < n) { \n\
    out[tid] = a * x[tid] + y[tid]; \n\
  } \n\
} \n";

const char *saxpy_r = R"(
extern "C" __global__ void saxpy_r(float a, float *x, float *y, float *out,
                                   size_t n) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = a * x[tid] + y[tid];
  }
};
)";

TEST(nvrtc, saxpy) {
  // Create an instance of hiprtcProgram
  hiprtcProgram prog;
  NVRTC_RESULT_CHECK(hiprtcCreateProgram(&prog,      // prog
                                        saxpy,      // code string
                                        "saxpy.cu", // name
                                        0,          // numHeaders
                                        NULL,       // headers
                                        NULL        // includeNames
                                        ));

  // Compile the program with fmad disabled.
  // Note: Can specify GPU target architecture explicitly with '-arch' flag.
  const char *opts[] = {"--fmad=false"};
  hiprtcResult compileResult = hiprtcCompileProgram(prog,  // prog
                                                  1,     // numOptions
                                                  opts); // options

  // Obtain compilation log from the program.
  size_t logSize;
  NVRTC_RESULT_CHECK(hiprtcGetProgramLogSize(prog, &logSize));
  std::string log(logSize, ' ');
  NVRTC_RESULT_CHECK(hiprtcGetProgramLog(prog, log.data()));
  std::cout << log << '\n';
  if (compileResult != HIPRTC_SUCCESS) {
    exit(1);
  }

  // Obtain PTX code from the program.
  size_t ptxSize;
  NVRTC_RESULT_CHECK(hiprtcGetCodeSize(prog, &ptxSize));
  std::string ptx(ptxSize, ' ');
  NVRTC_RESULT_CHECK(hiprtcGetCode(prog, ptx.data()));
  // std::cout << ptx << '\n';

  // Init cuda driver API
  hipDevice_t cuDevice;
  hipCtx_t context;
  CUDA_RESULT_CHECK(hipInit(0));
  CUDA_RESULT_CHECK(hipDeviceGet(&cuDevice, 0));
  CUDA_RESULT_CHECK(hipCtxCreate(&context, 0, cuDevice));

  // Load the generated PTX and get a handle to kernel.
  hipModule_t module;
  hipFunction_t kernel;
  CUDA_RESULT_CHECK(hipModuleLoadDataEx(&module, ptx.data(), 0, 0, 0));
  CUDA_RESULT_CHECK(hipModuleGetFunction(&kernel, module, "saxpy"));

  // Generate input for execution, and create output buffers.
  size_t n = NUM_THREADS * NUM_BLOCKS;
  size_t bufferSize = n * sizeof(float);
  float a = 5.1f;
  float *hX = new float[n], *hY = new float[n], *hOut = new float[n];
  for (size_t i = 0; i < n; ++i) {
    hX[i] = static_cast<float>(i);
    hY[i] = static_cast<float>(i * 2);
  }
  hipDeviceptr_t dX, dY, dOut;
  CUDA_RESULT_CHECK(hipMalloc(&dX, bufferSize));
  CUDA_RESULT_CHECK(hipMalloc(&dY, bufferSize));
  CUDA_RESULT_CHECK(hipMalloc(&dOut, bufferSize));
  CUDA_RESULT_CHECK(hipMemcpyHtoD(dX, hX, bufferSize));
  CUDA_RESULT_CHECK(hipMemcpyHtoD(dY, hY, bufferSize));

  // Execute SAXPY.
  void *args[] = {&a, &dX, &dY, &dOut, &n};
  CUDA_RESULT_CHECK(hipModuleLaunchKernel(kernel, NUM_BLOCKS, 1, 1, // grid dim
                                   NUM_THREADS, 1, 1,        // block dim
                                   0, NULL, // shared mem and stream
                                   args, 0));

  // arguments
  CUDA_RESULT_CHECK(hipCtxSynchronize());
  // Retrieve and print output.
  CUDA_RESULT_CHECK(hipMemcpyDtoH(hOut, dOut, bufferSize));

  for (size_t i = 0; i < n; ++i) {
    std::cout << a << " * " << hX[i] << " + " << hY[i] << " = " << hOut[i]
              << '\n';
  }

  // Release resources.
  CUDA_RESULT_CHECK(hipFree(dX));
  CUDA_RESULT_CHECK(hipFree(dY));
  CUDA_RESULT_CHECK(hipFree(dOut));
  CUDA_RESULT_CHECK(hipModuleUnload(module));
  CUDA_RESULT_CHECK(hipCtxDestroy(context));
  delete[] hX;
  delete[] hY;
  delete[] hOut;
  // Destroy the program.
  NVRTC_RESULT_CHECK(hiprtcDestroyProgram(&prog));
}