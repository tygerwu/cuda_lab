//---------- x.cu ----------

#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 8

extern __device__ int g[N];
extern __device__ void bar(void);

__global__ void foo(void) {

  __shared__ int a[N];
  a[threadIdx.x] = threadIdx.x;

  __syncthreads();

  g[threadIdx.x] = a[blockDim.x - threadIdx.x - 1];

  bar();
}

int Run() {
  unsigned int i;
  int *dg, hg[N];
  int sum = 0;

  foo<<<1, N>>>();

  if (hipGetSymbolAddress((void **)&dg, HIP_SYMBOL(g))) {
    printf("couldn't get the symbol addr\n");
    return 1;
  }
  if (hipMemcpy(hg, dg, N * sizeof(int), hipMemcpyDeviceToHost)) {
    printf("couldn't memcpy\n");
    return 1;
  }

  for (i = 0; i < N; i++) {
    sum += hg[i];
  }
  if (sum == 36) {
    printf("PASSED\n");
  } else {
    printf("FAILED (%d)\n", sum);
  }
  return 0;
}
